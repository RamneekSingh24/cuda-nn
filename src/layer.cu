#include "hip/hip_runtime.h"

#include "layer.cuh"
#include "cudaAssert.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <>
#include <math.h>
#include <cstdio>

CudaNNLayer::CudaNNLayer(int inputSize, int outputSize, float *values, float *na_values,
                         float *value_gradients, float *weights, float *weight_gradients,
                         float *biases, float *bias_gradients, NeuronActivation activationFunction)
    : numNeurons(outputSize), inputSize(inputSize), outputSize(outputSize),
      values(values), non_activated_values(na_values), value_gradients(value_gradients),
      weights(weights), weight_gradients(weight_gradients), biases(biases),
      bias_gradients(bias_gradients), activationFunction(activationFunction){};

__global__ void print_layer(float *input, float *output, float *weights,
                            float *biases, int inputSize, int outputSize, int batchSize, NeuronActivation activation)
{
    printf("\n-----------------\n");
    printf("\ninput\n");

    for (int i = 0; i < inputSize * batchSize; i++)
    {
        printf("%f ", input[i]);
    }
    printf("\noutput\n");
    for (int i = 0; i < outputSize * batchSize; i++)
    {
        printf("%f ", output[i]);
    }

    printf("\nweights\n");
    for (int i = 0; i < inputSize * outputSize; i++)
    {
        printf("%f ", weights[i]);
    }

    printf("\nbiases\n");
    for (int i = 0; i < outputSize; i++)
    {
        printf("%f ", biases[i]);
    }

    printf("\n-----------------\n");
}

__device__ __forceinline__ double sigmoid(float x)
{
    return 1.0 / (1.0 + exp(-x));
}

__global__ void forwardPropKernel(float *input, float *output, float *output_nact, float *weights,
                                  float *biases, int inputSize, int outputSize, int batchSize, NeuronActivation activation)
{
    // weights -> outputSize x inputSize
    // input -> inputSize x batchSize
    // output -> outputSize x batchSize
    // baises -> outputSize x 1

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // x'th value of the y'th data item
    int x = idx / batchSize;
    int y = idx % batchSize;

    // compute output[x][y]
    if (x < outputSize && y < batchSize)
    {
        float value = 0.0;
        for (int i = 0; i < inputSize; i++)
        {
            value += weights[index(x, i, inputSize)] * input[index(i, y, batchSize)];
        }
        value += biases[x];
        output_nact[index(x, y, batchSize)] = value;
        if (activation == Sigmoid)
        {
            value = sigmoid(value);
        }
        output[index(x, y, batchSize)] = value;
    }
}

void CudaNNLayer::forwardProp(float *input)
{
    // weights -> outputSize x inputSize
    // input -> inputSize x batchSize
    // output -> outputSize x batchSize
    // baises -> outputSize x 1

    // print_layer<<<1, 1>>>(input, values, weights, biases,
    //   inputSize, outputSize, BATCH_SIZE, activationFunction);

    dim3 blockSize(THREADS_PER_BLOCK);
    dim3 gridSize((outputSize * BATCH_SIZE + blockSize.x - 1) / blockSize.x);
    forwardPropKernel<<<blockSize, gridSize>>>(input, values, non_activated_values, weights, biases,
                                               inputSize, outputSize, BATCH_SIZE, activationFunction);
    // print_layer<<<1, 1>>>(input, values, weights, biases,
    //   inputSize, outputSize, BATCH_SIZE, activationFunction);

    cudaCheckError(hipDeviceSynchronize());
}

__device__ float activationDerivative(float x, NeuronActivation activation)
{
    if (activation == Identity)
    {
        return 1.0;
    }
    else if (activation == Sigmoid)
    {
        return sigmoid(x) * (1 - sigmoid(x));
    }
    return 1.0;
}

__global__ void backwardPropKernel(float *input_grad_vals, float *output_val_grads,
                                   float *neuron_vals_nact, float *weights,
                                   int inputSize, int outputSize,
                                   int batchSize, NeuronActivation activation)
{
    // input : outputSize x BATCH_SIZE
    // w transpose: inputSize x outputSize
    // out = w tranpose * input * scalar = inputSize x BATCH_SIZE

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // x'th value gradient of the y'th data item
    int x = idx / batchSize;
    int y = idx % batchSize;

    // compute output_val_grads[x][y]
    if (x < inputSize && y < batchSize)
    {
        float value = 0.0;
        // w transpose . input
        for (int i = 0; i < outputSize; i++)
        {
            // w transp(x, i) * input(i, y)
            // w(i, x) * input(i, y) * f' (non_activated(i, y))
            value += weights[index(i, x, inputSize)] *
                     input_grad_vals[index(i, y, batchSize)] *
                     activationDerivative(neuron_vals_nact[index(i, y, batchSize)], activation);
        }

        output_val_grads[index(x, y, batchSize)] = value;
    }
}

void CudaNNLayer::backwardProp(float *input)
{
    // input : outputSize x BATCH_SIZE
    // w transpose: inputSize x outputSize
    // out = w tranpose * input *elementwise f'(non_activated) = inputSize x BATCH_SIZE
    // weight gradients = input * values transpose(inp of fwd prop transpose)
    //                  = outputSize  x inputSize
    // bias = sum(input's col) = outputSize x 1
    dim3 blockSize(THREADS_PER_BLOCK);
    dim3 gridSize((inputSize * BATCH_SIZE + blockSize.x - 1) / blockSize.x);
    backwardPropKernel<<<blockSize, gridSize>>>(input, value_gradients,
                                                non_activated_values, weights,
                                                inputSize, outputSize,
                                                BATCH_SIZE, activationFunction);
    cudaCheckError(hipDeviceSynchronize());
}

__global__ void updateWeightsKernel(float *input_grad_vals, float *input_fwd_prop_input_vals,
                                    float *neuron_vals_nact, float *weights,
                                    float *biases, int inputSize, int outputSize,
                                    int batchSize, NeuronActivation activation)
{
    // input : outputSize x BATCH_SIZE
    // pwd_prop_inp_values transpose = BATCH_SIZE x inputSize
    // weight gradients = input_gradient * fwd_prop_inp_values transpose(inp of fwd prop transpose)
    //                  = outputSize  x inputSize
    // bias = sum(input's col) = outputSize x 1

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // (x,y) th weight gradient
    // weight connecting xth neuron and yth neuron in the prev layer
    int x = idx / inputSize;
    int y = idx % inputSize;

    // compute weight[x][y]
    if (x < outputSize && y < inputSize)
    {
        float w_gradient = 0.0;
        float bias_gradient = 0.0;
        // input . fwd_prop_inp_values_graidents
        for (int i = 0; i < BATCH_SIZE; i++)
        {
            // input_grad (x, i) * fwd_prop_inp transp (i, y)
            // input_grad(x, i) * f' (non_activated(x, i)) * fwd_prop_inp(y, i)
            w_gradient += (input_grad_vals[index(x, i, batchSize)] *
                           activationDerivative(neuron_vals_nact[index(x, i, batchSize)], activation) *
                           input_fwd_prop_input_vals[index(y, i, inputSize)]);
            bias_gradient += input_grad_vals[index(x, i, batchSize)] *
                             activationDerivative(neuron_vals_nact[index(x, i, batchSize)], activation);
        }
        w_gradient /= batchSize;
        bias_gradient /= batchSize;

        weights[index(x, y, inputSize)] -= LEARNING_RATE * w_gradient;
        if (y == 0)
        {
            biases[x] -= LEARNING_RATE * bias_gradient;
        }
    }
}

void CudaNNLayer::updateWeights(float *input_val_gradients, float *input_fwd_prop_input_vals)
{
    // input : outputSize x BATCH_SIZE
    // w transpose: inputSize x outputSize
    // out = w tranpose * input * scalar = inputSize x BATCH_SIZE
    // weight gradients = input * values transpose(inp of fwd prop transpose)
    //                  = outputSize  x inputSize
    // bias = sum(input's col) = outputSize x 1
    dim3 blockSize(THREADS_PER_BLOCK);
    dim3 gridSize((outputSize * inputSize + blockSize.x - 1) / blockSize.x);
    updateWeightsKernel<<<blockSize, gridSize>>>(input_val_gradients, input_fwd_prop_input_vals,
                                                 non_activated_values, weights,
                                                 biases, inputSize, outputSize,
                                                 BATCH_SIZE, activationFunction);
    cudaCheckError(hipDeviceSynchronize());
}
